#include "hip/hip_runtime.h"
#include "multi_padded_cast_transpose.h"

#include "utils.cuh"

namespace fp8_gmm {

namespace {

// Parameters to tune
constexpr int kNumWarpsPerTile = 4;
constexpr int kNumThreadsPerBlock = THREADS_PER_WARP * kNumWarpsPerTile;
constexpr int kDesiredLoadSize = 8;
constexpr int kDesiredStoreSize = 8;
constexpr int kMaxTensorsPerKernel = 32;  // Args must be <4 KB

struct MultiPaddedCastTransposeArgs {
  void* input_list[kMaxTensorsPerKernel];
  void* output_list[kMaxTensorsPerKernel];
  void* transpose_list[kMaxTensorsPerKernel];
  void* scale_list[kMaxTensorsPerKernel];
  void* amax_list[kMaxTensorsPerKernel];
  int num_rows_list[kMaxTensorsPerKernel];
  int row_length_list[kMaxTensorsPerKernel];
  int padded_rows_list[kMaxTensorsPerKernel];
  int block_range[kMaxTensorsPerKernel + 1];
  int num_tensors;
};

template <int nvec_in, int nvec_out, bool aligned, typename OType>
__global__ void __launch_bounds__(kNumThreadsPerBlock)
    multi_padded_cast_transpose_kernel(MultiPaddedCastTransposeArgs args) {
  using IVec = Vec<bf16, nvec_in>;
  using OVecC = Vec<OType, nvec_in>;
  using OVecT = Vec<OType, nvec_out>;

  // Thread indices
  // Note: Block is interpreted as a warp_size x num_warps grid
  constexpr int bdimx = THREADS_PER_WARP;
  constexpr int bdimy = kNumWarpsPerTile;
  const int tid = threadIdx.x;
  const int tidx = tid % bdimx;
  const int tidy = tid / bdimx;
  const int bid = blockIdx.x;

  // Input tensors are divided into tiles
  // Note: Each tile is a warp_size x warp_size grid of nvec_out x nvec_in subtiles
  constexpr int tile_dim_m = THREADS_PER_WARP * nvec_out;
  constexpr int tile_dim_n = THREADS_PER_WARP * nvec_in;

  // Number of nvec_out x nvec_in subtiles for each thread to
  // load/store
  constexpr int n_iterations = THREADS_PER_WARP / kNumWarpsPerTile;

  // Find tensor corresponding to block
  int tensor_id = 0;
  while (args.block_range[tensor_id + 1] <= bid) {
    ++tensor_id;
  }
  const bf16* input = reinterpret_cast<const bf16*>(args.input_list[tensor_id]);
  OType* output = reinterpret_cast<OType*>(args.output_list[tensor_id]);
  OType* transpose = reinterpret_cast<OType*>(args.transpose_list[tensor_id]);
  const fp32* scale_ptr = reinterpret_cast<fp32*>(args.scale_list[tensor_id]);
  const fp32 scale = scale_ptr == nullptr ? 1 : *scale_ptr;
  fp32* amax = reinterpret_cast<fp32*>(args.amax_list[tensor_id]);
  const int num_rows = args.num_rows_list[tensor_id];
  const int row_length = args.row_length_list[tensor_id];
  const int padded_rows = args.padded_rows_list[tensor_id];

  // Find position of tile within tensor
  const int num_tiles_n = (row_length + tile_dim_n - 1) / tile_dim_n;
  const int tile_id = bid - args.block_range[tensor_id];
  const int tile_id_m = tile_id / num_tiles_n;
  const int tile_id_n = tile_id % num_tiles_n;
  const int tile_row = tile_id_m * tile_dim_m;
  const int tile_col = tile_id_n * tile_dim_n;

  // Load input and store to registers
  // Note: Each thread loads n_iterations subtiles, casts to output
  // type, and transposes in registers.
  OVecT local_transpose[nvec_in][n_iterations];
  fp32 local_amax = 0;
#pragma unroll
  for (int iter = 0; iter < n_iterations; ++iter) {
    const int i1 = tidy + iter * bdimy;
    const int j1 = tidx;
#pragma unroll
    for (int i2 = 0; i2 < nvec_out; ++i2) {
      const int row = tile_row + i1 * nvec_out + i2;
      const int col = tile_col + j1 * nvec_in;
      IVec local_input;
      OVecC local_output;
      if constexpr (aligned) {
        local_input.load_from(&input[row * row_length + col]);
      } else {
        local_input.clear();
        if (row < num_rows) {
#pragma unroll
          for (int j2 = 0; j2 < nvec_in; ++j2) {
            if (col + j2 < row_length) {
              local_input.data.elt[j2] = input[row * row_length + col + j2];
            }
          }
        }
      }
#pragma unroll
      for (int j2 = 0; j2 < nvec_in; ++j2) {
        const fp32 x = fp32(local_input.data.elt[j2]);
        const OType y = OType(scale * x);
        local_output.data.elt[j2] = y;
        local_transpose[j2][iter].data.elt[i2] = y;
        local_amax = fmaxf(fabsf(x), local_amax);
      }
      if constexpr (aligned) {
        local_output.store_to(&output[row * row_length + col]);
      } else {
        if (row < num_rows) {
#pragma unroll
          for (int j2 = 0; j2 < nvec_in; ++j2) {
            if (col + j2 < row_length) {
              output[row * row_length + col + j2] = local_output.data.elt[j2];
            }
          }
        }
      }
    }
  }

  // Copy transposed output from registers to global memory
  __shared__ OVecT shared_transpose[THREADS_PER_WARP][THREADS_PER_WARP + 1];
#pragma unroll
  for (int j2 = 0; j2 < nvec_in; ++j2) {
#pragma unroll
    for (int iter = 0; iter < n_iterations; ++iter) {
      const int i1 = tidy + iter * bdimy;
      const int j1 = tidx;
      shared_transpose[j1][i1] = local_transpose[j2][iter];
    }
    __syncthreads();
#pragma unroll
    for (int iter = 0; iter < n_iterations; ++iter) {
      const int i1 = tidx;
      const int j1 = tidy + iter * bdimy;
      const int row = tile_row + i1 * nvec_out;
      const int col = tile_col + j1 * nvec_in + j2;
      if constexpr (aligned) {
        shared_transpose[j1][i1].store_to(&transpose[col * padded_rows + row]);
      } else {
        if (col < row_length) {
#pragma unroll
          for (int i2 = 0; i2 < nvec_out; ++i2) {
            if (row + i2 < padded_rows) {
              transpose[col * padded_rows + row + i2] = shared_transpose[j1][i1].data.elt[i2];
            }
          }
        }
      }
    }
    __syncthreads();
  }

  // Finalize fp8 factors
  local_amax = reduce_max<kNumWarpsPerTile>(local_amax, tidy);
  if (tid == 0) {
    static_assert(std::is_same<fp32, float>::value);
    if (amax != nullptr) atomicMaxFloat(amax, local_amax);
  }
}

}  // namespace

void MultiPaddedCastTranspose(std::vector<at::Tensor> input_list, std::vector<at::Tensor> output_list,
                              std::vector<at::Tensor> transpose_list, std::vector<at::Tensor> scale_list,
                              std::vector<at::Tensor> amax_list) {
  if (input_list.empty()) {
    return;
  }

  TORCH_CHECK(input_list.size() <= kMaxTensorsPerKernel);

  // Input matrices are divided into tiles
  // Note: Each tile is a warp_size x warp_size grid of nvec_out x nvec_in subtiles
  const int tile_dim_m = THREADS_PER_WARP * kDesiredStoreSize / sizeof(fp8e4m3);
  const int tile_dim_n = THREADS_PER_WARP * kDesiredLoadSize / sizeof(bf16);

  // Add tensors to kernel argument struct
  MultiPaddedCastTransposeArgs kernel_args_aligned, kernel_args_unaligned;
  kernel_args_aligned.num_tensors = 0;
  kernel_args_aligned.block_range[0] = 0;
  kernel_args_unaligned.num_tensors = 0;
  kernel_args_unaligned.block_range[0] = 0;
  for (size_t tensor_id = 0; tensor_id < input_list.size(); ++tensor_id) {
    TORCH_CHECK(input_list[tensor_id].scalar_type() == torch::kBFloat16);
    TORCH_CHECK(output_list[tensor_id].scalar_type() == torch::kFloat8_e4m3fn ||
                output_list[tensor_id].scalar_type() == torch::kFloat8_e5m2);
    TORCH_CHECK(output_list[tensor_id].scalar_type() == transpose_list[tensor_id].scalar_type());
    TORCH_CHECK(input_list[tensor_id].sizes() == output_list[tensor_id].sizes());
    const int num_rows = input_list[tensor_id].size(0);
    const int row_length = input_list[tensor_id].size(1);
    const int trans_rows = transpose_list[tensor_id].size(0);
    const int trans_cols = transpose_list[tensor_id].size(1);
    TORCH_CHECK(row_length == trans_rows && trans_cols >= num_rows);
    const int num_tiles_m = (trans_cols + tile_dim_m - 1) / tile_dim_m;
    const int num_tiles_n = (row_length + tile_dim_n - 1) / tile_dim_n;
    const int num_tiles = num_tiles_m * num_tiles_n;

    // Figure out whether to use aligned or unaligned kernel
    const bool aligned = ((trans_cols == num_rows) && (num_tiles_m * tile_dim_m == trans_cols) &&
                          (num_tiles_n * tile_dim_n == row_length));
    auto& kernel_args = aligned ? kernel_args_aligned : kernel_args_unaligned;

    // Add tensor to kernel argument struct
    const int pos = kernel_args.num_tensors;
    kernel_args.input_list[pos] = input_list[tensor_id].data_ptr();
    kernel_args.output_list[pos] = output_list[tensor_id].data_ptr();
    kernel_args.transpose_list[pos] = transpose_list[tensor_id].data_ptr();
    kernel_args.scale_list[pos] = scale_list[tensor_id].data_ptr();
    kernel_args.amax_list[pos] = amax_list[tensor_id].data_ptr();
    kernel_args.num_rows_list[pos] = num_rows;
    kernel_args.row_length_list[pos] = row_length;
    kernel_args.padded_rows_list[pos] = trans_cols;
    kernel_args.block_range[pos + 1] = kernel_args.block_range[pos] + num_tiles;
    kernel_args.num_tensors++;
  }

  // Launch kernel
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  constexpr int nvec_in = kDesiredLoadSize / sizeof(bf16);
  constexpr int nvec_out = kDesiredStoreSize / sizeof(fp8e4m3);
  if (kernel_args_aligned.num_tensors > 0) {
    if (output_list[0].scalar_type() == torch::kFloat8_e4m3fn) {
      multi_padded_cast_transpose_kernel<nvec_in, nvec_out, true, fp8e4m3>
          <<<kernel_args_aligned.block_range[kernel_args_aligned.num_tensors], kNumThreadsPerBlock, 0, stream>>>(
              kernel_args_aligned);
    } else {
      multi_padded_cast_transpose_kernel<nvec_in, nvec_out, true, fp8e5m2>
          <<<kernel_args_aligned.block_range[kernel_args_aligned.num_tensors], kNumThreadsPerBlock, 0, stream>>>(
              kernel_args_aligned);
    }
  }
  if (kernel_args_unaligned.num_tensors > 0) {
    if (output_list[0].scalar_type() == torch::kFloat8_e4m3fn) {
      multi_padded_cast_transpose_kernel<nvec_in, nvec_out, false, fp8e4m3>
          <<<kernel_args_unaligned.block_range[kernel_args_unaligned.num_tensors], kNumThreadsPerBlock, 0, stream>>>(
              kernel_args_unaligned);
    } else {
      multi_padded_cast_transpose_kernel<nvec_in, nvec_out, false, fp8e5m2>
          <<<kernel_args_unaligned.block_range[kernel_args_unaligned.num_tensors], kNumThreadsPerBlock, 0, stream>>>(
              kernel_args_unaligned);
    }
  }
}

}  // namespace fp8_gmm
